#include <hip/hip_runtime.h>


#define imin(a, b) (a < b ? a : b)
const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void cudadot(float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while(tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while(i != 0)
    {
        if(cacheIndex < i)
        {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    if(cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

void dot(float *a, float *b, float *c)
{
    cudadot<<<blocksPerGrid, threadsPerBlock>>>(a, b, c);
}
