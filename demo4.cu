#include <hip/hip_runtime.h>


#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

void cudaAdd(int *a, int *b, int *c)
{
    add<<<128, 128>>>(a, b, c);
}
