#include <hip/hip_runtime.h>


#define N 10

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x;
    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

void cudaAdd(int *a, int *b, int *c)
{
    add<<<1, N>>>(a, b, c);
}
