#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if(tid < 10)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void cudaAdd(int *a, int *b, int *c)
{
    add<<<10, 1>>>(a, b, c);
}
